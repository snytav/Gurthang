#include "hip/hip_runtime.h"


#ifdef __HIPCC__
 __host__ __device__
 #endif
#ifdef VIRTUAL_FUNCTIONS
virtual
#endif
void Move(unsigned int i,int *cells,CurrentTensor *t1,CurrentTensor *t2,double mass,double q_mass,
		 CellDouble *Ex1,CellDouble *Ey1,CellDouble *Ez1,
		 CellDouble *Hx1,CellDouble *Hy1,CellDouble *Hz1)
{
     double3 x,x1;
     double  m,q_m;
     Particle p;
     Field fd;

     if(i >= number_of_particles) return;
     p = readParticleFromSurfaceDevice(i);
	 fd = GetField(&p,Ex1,Ey1,Ez1,Hx1,Hy1,Hz1);

	 p.Move(fd.E,fd.H,tau);
	 writeParticleToSurface(i,&p);

	 Particle p1;

}

#ifdef __HIPCC__
 __host__ __device__
 #endif
 void AccCurrent(unsigned int i,int *cells,CurrentTensor *t1,CurrentTensor *t2,double mass,double q_mass)
 {
	 Particle p;
	 if(i >= number_of_particles) return;

	 p = readParticleFromSurfaceDevice(i);
	 CurrentToMesh(tau,cells,t1,t2,&p);

     writeParticleToSurface(i,&p);
}
