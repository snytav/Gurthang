#include "hip/hip_runtime.h"






template <template <class Particle> class Cell >
global_for_CUDA
void printParticle(Cell<Particle>  **cells,int num,int sort)
{
	unsigned int nx = blockIdx.x;
	unsigned int ny = blockIdx.y;
	unsigned int nz = blockIdx.z;
//	int i,l,k;
	Cell<Particle>  *c,*c0 = cells[0],nc;
	//double t;
	__shared__ extern CellDouble fd[9];
	//double *src,*dst;
	//int pqr2;
	//CurrentTensor t1,t2;
	Particle p;

	c = cells[ c0->getGlobalCellNumber(nx,ny,nz)];
//	c = cells[ n ];

	nc = *c;
    if(nc.number_of_particles < threadIdx.x) return;

	nc.readParticleFromSurfaceDevice(threadIdx.x,&p);

		if(p.fortran_number == num && (int)p.sort == sort)
		{
//#ifdef PARTICLE_CELL_DEBUG_PRINTS
			printf("particle-print %5d thread %3d cell (%d,%d,%d) sort %d  %25.15e,%25.15e,%25.15e \n",p.fortran_number,threadIdx.x,c->i,c->l,c->k,(int)p.sort,p.x,p.y,p.z);
//#endif
		}
}














template <template <class Particle> class Cell >
global_for_CUDA void GPU_WriteControlSystem(Cell<Particle>  **cells)
{
	unsigned int nx = blockIdx.x;
	unsigned int ny = blockIdx.y;
	unsigned int nz = blockIdx.z;
//	int i,l,k;
	Cell<Particle>  *c,*c0 = cells[0],nc;
	//double t;
	__shared__ extern CellDouble fd[9];
	//double *src; //,*dst;
//	int pqr2;
	//CurrentTensor t1,t2;

	c = cells[ c0->getGlobalCellNumber(nx,ny,nz)];
//	c = cells[ n ];

	 nc = *c;

	 nc.SetControlSystemToParticles();

}

//TODO : 1. 3 separate kernels :
//            A. form 3x3x3 array with number how many to fly and departure list with start positions in 3x3x3 array
//            B. func to get 3x3x3 indexes from a pair of cell numbers, to and from function
//            C. 2nd kernel to write arrival 3x3x3 matrices
///           D. 3rd kernel to form arrival positions in the particle list
//            E. 4th to write arriving particles




























