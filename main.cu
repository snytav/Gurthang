#include "gpu_plasma.h"
#include <stdlib.h>
#include "mpi_shortcut.h"
//TODO: gpu cell in the global array at copy from there appears to be not initialized

int main(int argc,char*argv[])
{
   GPUPlasma<GPUCell> *plasma;

   plasma = new GPUPlasma<GPUCell>(100,4,4,1.1424,0.05,0.05,1.0,2000,1.0,0.001);

   plasma->Initialize();

   double t = plasma->compareCPUtoGPU();
   printf("----------------------------------------------------------- plasma check before move %.5f\n",t);
   size_t m_free,m_total;

   hipMemGetInfo(&m_free,&m_total);

   struct sysinfo info;


   for(int nt = START_STEP_NUMBER;nt <= TOTAL_STEPS;nt++)
   {
	   hipMemGetInfo(&m_free,&m_total);
	   sysinfo(&info);
#ifdef MEMORY_PRINTS
       printf("before Step  %10d CPU memory free %10u GPU memory total %10d free %10d\n",
    		   nt,info.freeram/1024/1024,m_total/1024/1024,m_free/1024/1024);
#endif

       plasma->Step(nt);

       hipMemGetInfo(&m_free,&m_total);
       sysinfo(&info);
#ifdef MEMORY_PRINTS
       printf("after  Step  %10d CPU memory free %10u GPU memory total %10d free %10d\n",
    		   nt,info.freeram/1024/1024/1024,m_total/1024/1024/1024,m_free/1024/1024/1024);
#endif
   }


   t = plasma->compareCPUtoGPU();
   printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ plasma check after move %.5f\n",t);

   delete plasma;
   

   return 0;
}
