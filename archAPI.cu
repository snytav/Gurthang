#include "hip/hip_runtime.h"
/*
 * archAPI.cxx
 *
 *  Created on: Apr 10, 2018
 *      Author: snytav
 */
#include<stdlib.h>
#include<string.h>


#include "archAPI.h"



#ifdef __HIPCC__
int SetDevice(int n){return hipSetDevice(n);}
#else
int SetDevice(int n){return 0;}
#endif

#ifdef __HIPCC__
__device__
void AsyncCopy(double *dst,double *src,int n,int size)
{
	int j;
	j = n;
	if(j < size)
	{
	   dst[j] = src[j];
	}

}
#else
void AsyncCopy(double *dst,double *src,int n,int size){ memcpy(dst,src,n,size);}
#endif


#ifdef __HIPCC__
 int MemoryCopy(void* dst,void *src,size_t size,int dir)
{
//	int err = 0;


	hipMemcpyKind cuda_dir;

	if(dir == HOST_TO_DEVICE) cuda_dir = hipMemcpyHostToDevice;
	if(dir == HOST_TO_HOST) cuda_dir = hipMemcpyHostToHost;
	if(dir == DEVICE_TO_HOST) cuda_dir = hipMemcpyDeviceToHost;
	if(dir == DEVICE_TO_DEVICE) cuda_dir = hipMemcpyDeviceToDevice;



	return ((int)hipMemcpy(dst,src,size,cuda_dir));
}
#else
 int MemoryCopy(void* dst,void *src,size_t size,int dir);
#endif


#ifdef __HIPCC__
 int MemoryAllocate(void** dst,size_t size)
{
	hipMalloc(dst,size);
    return 0;
}
#else
 int MemoryAllocate(void** dst,size_t size);
#endif

#ifndef __HIPCC__
int GetDeviceMemory(size_t *m_free,size_t *m_total)
{
	*m_free = 0;
	*m_total = 0;
	return 0;
}
#endif

#ifndef __HIPCC__
int MemorySet(void *s, int c, size_t n)
{
	memset(s,c,n);
    return 0;
}
#endif


#ifndef __HIPCC__
int DeviceSynchronize()
{
    return 0;
}

#ifdef __HIPCC__
 int __host__ ThreadSynchronize()
{
	 return hipDeviceSynchronize();
}
#else
 int ThreadSynchronize()
 {
 	 return 0;
 }
#endif

 int getLastError()
{
	return 0;
}
#else
int getLastError()
{
	return (int)hipGetLastError();
}
#endif

#ifdef __HIPCC__
 __device__ void BlockThreadSynchronize()
 {
     __syncthreads();
 }
#else
 void BlockThreadSynchronize(){}
#endif

#ifdef __HIPCC__
__device__ double MultiThreadAdd(double *address, double val)
{
    double assumed,old=*address;
    do {
        assumed=old;
        old= __longlong_as_double(atomicCAS((unsigned long long int*)address,
                    __double_as_longlong(assumed),
                    __double_as_longlong(val+assumed)));
    }while (assumed!=old);

    *address += val;

    old = *address;

    return old;
}
#else
double MultiThreadAdd(double *address, double val){
#pragma omp critical
	*address += val;

}
#endif

#ifdef __HIPCC__
 const char *getErrorString(int err)
{
	return hipGetErrorString((hipError_t)err);
}
#else
const char *getErrorString(int err){return "";}

#endif


#ifdef __HIPCC__
 int GetDeviceMemory(size_t *m_free,size_t *m_total)
{
	return hipMemGetInfo(m_free,m_total);
}
#else
 int GetDeviceMemory(size_t *m_free,size_t *m_total){*m_free = -1; *m_total = -1;}
#endif


#ifdef __HIPCC__
int MemorySet(void *s, int c, size_t n)
{
	return (int)hipMemset(s,c,n);

//    return 0;
}
#else
int MemorySet(void *s, int c, size_t n)
{
	return memset(s,c,n);
}
#endif
